#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void vect_add(int *A, int *B, int *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    int n = 1000000;
    int size = n * sizeof(int);

    int *A, *B, *C;
    int *Dev_A, *Dev_B, *Dev_C;

    hipHostMalloc(&A, size, hipHostMallocDefault);
    hipHostMalloc(&B, size, hipHostMallocDefault);
    hipHostMalloc(&C, size, hipHostMallocDefault);

    for (int i = 0; i < n; i++)
    {
        A[i] = i;
        B[i] = i * 2;
    }

    hipMalloc(&Dev_A, size);
    hipMalloc(&Dev_B, size);
    hipMalloc(&Dev_C, size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipMemcpy(Dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(Dev_B, B, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlock = (n + blockSize - 1) / blockSize;

    vect_add<<<numBlock, blockSize>>>(Dev_A, Dev_B, Dev_C, n);

    hipMemcpy(C, Dev_C, size, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    for (int i = 0; i < 10; i++)
        cout << C[i] << " ";
    cout << endl;
    cout << "Time Taken: " << ms / 1000 << endl;

    hipFree(Dev_A);
    hipFree(Dev_B);
    hipFree(Dev_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}